#include "hip/hip_runtime.h"
#include "photon.h"
#include <iostream>

__global__ void normalisePhotonVelocities(Photon photons[], int width, int height)
{
    int photon_num = blockDim.x * blockIdx.x + threadIdx.x;
    if (photon_num < width*height)
    {
        photons[photon_num].v[1] = 1.;
        photons[photon_num].v[2] = 1.;
        photons[photon_num].v[3] = 1.;
        photons[photon_num].setMetric();
        photons[photon_num].makeVNull();
    }
}

int main()
{
    // CUDA photon allocation test.
    hipError_t err { hipSuccess };

    int width { 2560 };
    int height { 1440 };
    const int num_photons { width*height };

    // Allocate memory on host.
    size_t photon_array_size { num_photons * sizeof(Photon) };
    Photon *host_photons { (Photon*)malloc(photon_array_size) };

    // Allocate device memory.
    Photon *device_photons { nullptr };
    err = hipMalloc((void **)&device_photons, photon_array_size);
    err = hipMemcpy(device_photons, host_photons, photon_array_size, hipMemcpyHostToDevice);

    // Call kernel.
    int threadsPerBlock { 256 };
    int numBlocks { (num_photons + threadsPerBlock - 1) / threadsPerBlock };
    normalisePhotonVelocities<<<numBlocks, threadsPerBlock>>>(device_photons, width, height);
    err = hipGetLastError();

    // Transfer back to host.
    err = hipMemcpy(host_photons, device_photons, photon_array_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        std::cout << "FAILED" << "\n";
    }

    std::cout << host_photons[0].v[0] << "\n";
    std::cout << host_photons[0].v[1] << "\n";
    std::cout << host_photons[0].v[2] << "\n";
    std::cout << host_photons[0].v[3] << "\n";

    // Free host memory.
    free(host_photons);

    // Free device memory.
    err = hipFree(device_photons);

    return 0;
}
