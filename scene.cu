#include "hip/hip_runtime.h"
#include "scene.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <stdexcept>

// These objects and functions are used by the GPU when tracing photons. They are defined outside Scene
// in a separate namespace so that the GPU doesn't need a copy of the entire Scene object in device memory when calculating paths.
namespace DeviceTraceTools
{
    // Device variables and functions.
    // This pointer is left in host memory and assigned with hipMemcpy because otherwise it cannot be deallocated by the host
    // with hipFree(). This points to the sky map in device memory.
    unsigned char *device_sky_map { nullptr };
    // This points to the pixel array for the camera in device memory.
    unsigned char *device_camera_pixel_array { nullptr };
    // Default camera coordinates are along -x and the camera faces along +x.
    __device__ float device_camera_coords[4];
    __device__ float device_camera_quat[4];
    __device__ int device_pixels_w;
    __device__ int device_pixels_h;
    __device__ float device_fov_conversion_factor;
    __device__ int device_sky_pixels_w;
    __device__ int device_sky_pixels_h;
    __device__ const float device_sky_map_distance_squared{ 50.*50. };

    __device__ void calculateStartVelocity(float pixel_x, float pixel_y, float photon_v[4], float metric[4][4]);
    __device__ void getMetricTensor(float x_func[4], float metric_func[4][4]);
    __device__ void getChristoffelSymbols(float x_func[4], float metric_func[4][4], float c_symbols_func[4][4][4], float metric_derivs[4][4][4]);
    __device__ void makeVNull(float v_func[4], float metric_func[4][4]);
    __device__ void normaliseV(float v_func[4]);
    __device__ void invertMetric(float metric_func[4][4], float metric_inverse[4][4]);
    __device__ float calculateParameterStep(float metric[4][4]);
    __device__ void advance(float x[4], float v[4], float metric[4][4], float c_symbols[4][4][4], float metric_derivs[4][4][4]);
    __device__ void readPixelFromSkyMap(unsigned char *pixel, unsigned char *device_sky_map, int &x, int &y, int &sky_pixels_w, int &byte_depth);
};

// Initialise Scene object with no sky map and default camera parameters.
void Scene::initialiseDefault(char sky_map[])
{
    // Note sky_map will already have decayed to a char* pointer here; no need to convert.
    importSkyMap(sky_map);
    // // Set camera quaternion to default position and orientation and copy to device.
    // setCameraCoordinates((float*)&default_camera_coords);
    // setCameraQuaternion((float*)&default_camera_quat);
    // // Default resolution of 1920x1080.
    // setCameraRes(default_width, default_height);
    // // Default horizontal FoV is 75 degrees.
    // setCameraFoV(default_fov);
}

// Sky map image should be a 2:1 aspect ratio, 360-degree panoramic image, but there is no restriction on this.
void Scene::importSkyMap(char image_path[])
{
    // image_path should be a pointer to a C-style array of char[].
    // This is usually too large for stack allocation, so host_sky_map becomes a pointer to a pixel array on the heap.
    // Force to load as RGB (3 bytes per pixel).
    host_sky_map = stbi_load(image_path, &sky_pixels_w, &sky_pixels_h, &byte_depth, 3);
    if (host_sky_map != NULL)
    {
        sky_pixels_w_f = static_cast<float>(sky_pixels_w);
        sky_pixels_h_f = static_cast<float>(sky_pixels_h);
        phi_interval = (2.*pi_host) / sky_pixels_w;
        theta_interval = pi_host / sky_pixels_h;

        // Reset existing device map (if it exists), then copy the new map and related information.
        // This cannot be allocated in initialiseDefault() because its size is only known at run time.
        hipFree(DeviceTraceTools::device_sky_map);
        hipError_t err { hipSuccess };
        size_t map_size { sizeof(unsigned char)*sky_pixels_w*sky_pixels_h*byte_depth };
        err = hipMalloc((void **)&DeviceTraceTools::device_sky_map, map_size);
        if (err != hipSuccess)
        {
            throw std::runtime_error("Error: failed to allocate memory for sky map on device.");
        }
        err = hipMemcpy(DeviceTraceTools::device_sky_map, host_sky_map, map_size, hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            throw std::runtime_error("Error: failed to copy sky map to device.");
        }

        err = hipMemcpyToSymbol(HIP_SYMBOL(DeviceTraceTools::device_sky_pixels_w), &sky_pixels_w, sizeof(int));
        if (err != hipSuccess)
        {
            throw std::runtime_error("Error: failed to copy sky pixel width to device.");
        }
        err = hipMemcpyToSymbol(HIP_SYMBOL(DeviceTraceTools::device_sky_pixels_h), &sky_pixels_h, sizeof(int));
        if (err != hipSuccess)
        {
            throw std::runtime_error("Error: failed to copy sky pixel height to device.");
        }
    }
    else
    {
        throw std::runtime_error("Error: failed to load sky map image.");
    }
}

void Scene::runTraceKernel()
{
    // Use 32 threads per block for now. This is mostly limited by the available shared memory to store the Christoffel symbols and metric derivatives.
    // Use smaller blocks also allows the scheduler to naturally load-balance against the fact that pixels looking into the black hole probably require
    // more computation.
    dim3 threadsPerBlock(8, 4);
    int num_blocks_x { pixels_w / 8 };
    int num_blocks_y { pixels_h / 4 };
    if (pixels_w % 8 > 0)
    {
        num_blocks_x += 1;
    }
    if (pixels_h % 4 > 0)
    {
        num_blocks_y += 1;
    }
    dim3 numBlocks(num_blocks_x, num_blocks_y);
    for (int i { 0 }; i < 50; i++)
    {
        traceImage<<<numBlocks, threadsPerBlock>>>(DeviceTraceTools::device_sky_map, DeviceTraceTools::device_camera_pixel_array);
    }
    // Copy image back to host.
    hipError_t err;
    err = hipMemcpy(host_camera_pixel_array, DeviceTraceTools::device_camera_pixel_array, 3*sizeof(unsigned char)*pixels_w*pixels_h, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        throw std::runtime_error("Error: failed to copy camera pixel array from device to host.");
    }
}

void Scene::writeCameraImage(char image_path[])
{
    unsigned char *data { &host_camera_pixel_array[0] };
    stbi_write_jpg(image_path, pixels_w, pixels_h, 3, data, 100);
}

void Scene::freeHostPixelArrays()
{
    stbi_image_free(host_sky_map);
    free(host_camera_pixel_array);
}

void Scene::freeDevicePixelArrays()
{
    hipFree(DeviceTraceTools::device_sky_map);
    hipFree(DeviceTraceTools::device_camera_pixel_array);
}

// Sets the width and height resolution of the camera and copies it to the device.
void Scene::setCameraRes(int width, int height)
{
    pixels_w = width;
    pixels_h = height;
    hipError_t err;
    err = hipMemcpyToSymbol(HIP_SYMBOL(DeviceTraceTools::device_pixels_w), &pixels_w, sizeof(int));
    if (err != hipSuccess)
    {
        throw std::runtime_error("Error: failed to copy camera pixel width to device.");
    }
    err = hipMemcpyToSymbol(HIP_SYMBOL(DeviceTraceTools::device_pixels_h), &pixels_h, sizeof(int));
    if (err != hipSuccess)
    {
        throw std::runtime_error("Error: failed to copy camera pixel height to device.");
    }
    // Allocate memory for the camera pixel array.
    free(host_camera_pixel_array);
    hipFree(DeviceTraceTools::device_camera_pixel_array);
    host_camera_pixel_array = (unsigned char*)malloc(3*sizeof(unsigned char)*pixels_w*pixels_h);
    if (host_camera_pixel_array == nullptr)
    {
        throw std::runtime_error("Error: failed to allocate memory for camera pixel array on host.");
    }
    err = hipMalloc((void **)&DeviceTraceTools::device_camera_pixel_array, 3*sizeof(unsigned char)*pixels_w*pixels_h);
    if (err != hipSuccess)
    {
        throw std::runtime_error("Error: failed to allocate memory for camera pixel array on device.");
    }
}

// Set a new FoV in degrees and transfer the corresponding conversion factor to the device.
void Scene::setCameraFoV(float new_fov_width)
{
    fov_width = new_fov_width;
    fov_width_rad = fov_width * (pi_host/180.f);
    float conversion_factor = fov_width_rad / pixels_w;
    hipError_t err;
    err = hipMemcpyToSymbol(HIP_SYMBOL(DeviceTraceTools::device_fov_conversion_factor), &conversion_factor, sizeof(float));
    if (err != hipSuccess)
    {
        throw std::runtime_error("Error: failed to copy FoV conversion factor to device.");
    }
}

// When called, this will both set the host variable and copy it to DeviceTraceTools::camera_coords
// to let the GPU access its own copy in device memory. Same thing for setCameraQuaternion.
void Scene::setCameraCoordinates(float x[4])
{
    for (int i { 0 }; i < 4; i++)
    {
        camera_coords[i] = x[i];
    }
    hipError_t err { hipMemcpyToSymbol(HIP_SYMBOL(DeviceTraceTools::device_camera_coords), camera_coords, sizeof(float)*4) };
    if (err != hipSuccess)
    {
        throw std::runtime_error("Error: failed to copy camera coordinates to device.");
    }
}

void Scene::setCameraQuaternion(float quaternion[4])
{
    for (int i { 0 }; i < 4; i++)
    {
        camera_quat[i] = quaternion[i];
    }
    hipError_t err { hipMemcpyToSymbol(HIP_SYMBOL(DeviceTraceTools::device_camera_quat), camera_quat, sizeof(float)*4) };
    if (err != hipSuccess)
    {
        throw std::runtime_error("Error: failed to copy camera quaternion to device.");
    }
}

// Calculates the start velocity of a photon at pixel (x, y), where (0, 0) is the top-left corner of the image.
__device__ void DeviceTraceTools::calculateStartVelocity(float pixel_x, float pixel_y, float photon_v[4], float metric[4][4])
{
    float phi { (pixel_x - 0.5f*device_pixels_w) * device_fov_conversion_factor };
    float theta { (pixel_y - 0.5f*device_pixels_h) * device_fov_conversion_factor + 0.5f*pi_device };
    // Convert to Cartesian coordinates.
    float unrotated_v[4];
    unrotated_v[0] = 0.;
    unrotated_v[1] = sin(theta)*cos(phi);
    unrotated_v[2] = sin(theta)*sin(phi);
    unrotated_v[3] = cos(theta);
    // Rotate to align with the camera orientation.
    rotateVecByQuat(unrotated_v, device_camera_quat, photon_v);
    // Set the t-component to make the velocity null.
    makeVNull(photon_v, metric);
}

// Currently defined to return the Schwarzschild metric with a Schwarzschild radius of 1.
// Gets the metric at x_func and overwrites it into metric_func.
__device__ void DeviceTraceTools::getMetricTensor(float x_func[4], float metric_func[4][4])
{
    const float r_s { 1. };
    float r { norm3df(x_func[1], x_func[2], x_func[3]) };
    float r_squared { r*r };
    float mult_factor { r_s / (r_squared*(r-r_s)) };
    for (int mu { 1 }; mu < 4; mu++)
    {
        metric_func[0][mu] = 0.;
        metric_func[mu][0] = 0.;
        for (int nu { mu }; nu < 4; nu++)
        {
            metric_func[mu][nu] = mult_factor*x_func[mu]*x_func[nu];
            metric_func[nu][mu] = metric_func[mu][nu];
        }
    }
    metric_func[0][0] = -1. + r_s/r;
    metric_func[1][1] += 1.;
    metric_func[2][2] += 1.;
    metric_func[3][3] += 1.;
}

__device__ void DeviceTraceTools::getChristoffelSymbols(float x_func[4], float metric_func[4][4], float c_symbols_func[4][4][4], float metric_derivs[4][4][4])
{
    // Assumed default step in each coordinate.
    // TODO: How do you define this adaptively to not break near areas of extreme distortion?
    // For now, just set it to a small number.
    const float step { 1e-4 };
    const float inverse_step { 1./step };

    // Simple Euler forward-difference derivatives of the metric along each component.
    float metric_temp[4][4];
    for (int alpha { 0 }; alpha < 4; alpha++)
    {
        // WARNING: Euler can be significantly faster than central difference; just use a very small step and it'll probably be okay.
        // The limiting factor in accuracy is probably the overarching parameter step in the geodesic equation, not the metric derivatives.
        // TODO: Automatic differentiation with dual numbers?
        x_func[alpha] += step;
        getMetricTensor(x_func, metric_temp);
        for (int mu { 0 }; mu < 4; mu++)
        {
            for (int nu { mu }; nu < 4; nu++)
            {
                metric_derivs[alpha][mu][nu] = (metric_temp[mu][nu] - metric_func[mu][nu])*inverse_step;
                metric_derivs[alpha][nu][mu] = metric_derivs[alpha][mu][nu];
            }
        }
        x_func[alpha] -= step;
    }

    // Calculate the inverse of metric_func and overwrite it into metric_temp.
    // Set metric_temp to the identity matrix first.
    for (int mu { 0 }; mu < 4; mu++)
    {
        for (int nu { mu }; nu < 4; nu++)
        {
            if (mu == nu)
            {
                metric_temp[mu][nu] = 1.;
            }
            else
            {
                metric_temp[mu][nu] = 0.;
                metric_temp[nu][mu] = 0.;
            }
        }
    }
    // Store the inverse metric into metric_temp. metric_func is now useless
    // until it is assigned again in getMetricTensor() (it gets overwritten by invertMetric()).
    invertMetric(metric_func, metric_temp);

    // Calculate the 40 independent Christoffel symbols.
    for (int alpha { 0 }; alpha < 4; alpha++)
    {
        for (int mu { 0 }; mu < 4; mu++)
        {
            for (int nu { mu }; nu < 4; nu++)
            {
                float component[4];
                #pragma unroll
                for (int gamma { 0 }; gamma < 4; gamma++)
                {
                    component[gamma] = metric_derivs[nu][mu][gamma] + metric_derivs[mu][nu][gamma] - metric_derivs[gamma][mu][nu];
                }
                // Remember that metric_temp is the inverse metric here.
                c_symbols_func[alpha][mu][nu] = 0.5*(
                    metric_temp[alpha][0]*component[0] + metric_temp[alpha][1]*component[1]
                    + metric_temp[alpha][2]*component[2] + metric_temp[alpha][3]*component[3]
                );
                c_symbols_func[alpha][nu][mu] = c_symbols_func[alpha][mu][nu];
            }
        }
    }
}

/*
 * Modifies the t-component of the 4-velocity to make the vector null.
 * This requires solving a quadratic equation for the t-component; assume
 * that you should take the positive root because a=g_00 is
 * probably negative. Note that the more negative solution is needed
 * because the raytracer evolves photons "backwards".
 */
__device__ void DeviceTraceTools::makeVNull(float v_func[4], float metric_func[4][4])
{
    float a { metric_func[0][0] };
    float b { 0. };
    // c is the scalar product of the spatial metric with the spatial velocity components.
    float c { 0. };
    float contraction;
    for (int i { 1 }; i < 4; i++)
    {
        b += metric_func[0][i] * v_func[i];
        contraction = 0.;
        for (int j { 1 }; j < 4; j++)
        {
            contraction += metric_func[i][j] * v_func[j];
        }
        c += v_func[i]*contraction;
    }
    b *= 2.;

    // Take the positive root solution (note a=g_00 is usually negative, so this normally gives the negative solution).
    v_func[0] = (-b + sqrt(b*b - 4.*a*c)) / (2.*a);
}

// Makes the L2 norm of the velocity 1 for the sake of maintaining a roughly consistent affine parameter.
// This does turn it into a "unit" velocity!
__device__ void DeviceTraceTools::normaliseV(float v_func[4])
{
    float inv_norm { rnorm4df(v_func[0], v_func[1], v_func[2], v_func[3]) };
    v_func[0] *= inv_norm;
    v_func[1] *= inv_norm;
    v_func[2] *= inv_norm;
    v_func[3] *= inv_norm;
}

// TODO: This doesn't get the correct result for asymmetric matrices! Not technically important here, but it's
// indicative that something is wrong underneath.
__device__ void DeviceTraceTools::invertMetric(float metric_func[4][4], float metric_inverse[4][4])
{
    // Assume that that there are no zeros on the diagonal of metric_func and that metric_inverse is currently the identity matrix.
    // Invert with forward and backward-propagation (i.e. LU-decomposition). metric_func and metric_temp are both overwritten to avoid memory allocation.
    // WARNING: For now, assume that there are no zeros on the diagonal of the metric (very unlikely in t, x, y, z coordinates).

    float multiplier;

    // Forward-propagation pass.
    for (int i { 0 }; i < 3; i++)
    {
        for (int j { i+1 }; j < 4; j++)
        {
            multiplier = metric_func[j][i] / metric_func[i][i];
            // Use the accumulating zeros in the lower-triangular half to reduce the number of calculations.
            for (int k { i }; k < 4; k++)
            {
                metric_func[j][k] -= multiplier*metric_func[i][k];
            }
            // Use the fact that metric_inverse is currently an identity matrix to reduce the number of calculations.
            for (int k { 0 }; k < j; k++)
            {
                metric_inverse[j][k] -= multiplier*metric_inverse[i][k];
            }
        }
    }

    // Backward-propagation pass.
    for (int i { 3 }; i > 0; i--)
    {
        for (int j { i-1 }; j > -1; j--)
        {
            multiplier = metric_func[j][i] / metric_func[i][i];
            // Use the zeros in the lower-triangular half of metric_func to reduce the number of calculations.
            for (int k { i }; k < 4; k++)
            {
                metric_func[j][k] -= multiplier*metric_func[i][k];
            }
            #pragma unroll
            for (int k { 0 }; k < 4; k++)
            {
                metric_inverse[j][k] -= multiplier*metric_inverse[i][k];
            }
        }
    }

    // Last task is to normalise the rows of metric_inverse by whatever is left in the diagonal of metric_func.
    for (int i { 0 }; i < 4; i++)
    {
        multiplier = 1./metric_func[i][i];
        metric_func[i][i] = 1.;
        #pragma unroll
        for (int k { 0 }; k < 4; k++)
        {
            metric_inverse[i][k] *= multiplier;
        }
    }
}

// Crude way of testing how distorted the metric is from the Minkowski metric without resorting to the Riemann tensor.
// Used for adaptive step size. This only works in (t, x, y, z) coordinates.
__device__ float DeviceTraceTools::calculateParameterStep(float metric[4][4])
{
    // "Normalise" against things that scale the whole metric but introduce no curvature.
    float scale_factor { 0. };
    #pragma unroll
    for (int i { 0 }; i < 4; i++)
    {
        scale_factor += fabs(metric[i][i]);
    }
    scale_factor *= 0.25;
    scale_factor = 1./scale_factor;

    // Subtract the Minkowski metric from the scaled metric and add up all the absolute components.
    float deviation { 0. };
    deviation += fabs(metric[0][0]*scale_factor + 1.);
    #pragma unroll
    for (int i { 1 }; i < 4; i++)
    {
        // Diagonal components.
        deviation += fabs(metric[i][i]*scale_factor - 1.);
    }
    for (int i { 0 }; i < 3; i++)
    {
        for (int j { i+1 }; j < 4; j++)
        {
            // Off-diagonal components.
            deviation += 2.*fabs(metric[i][j]*scale_factor);
        }
    }

    if (deviation == 0)
    {
        // Metric is flat; set to the maximum step size.
        return 5.;
    }
    else
    {
        float dl;
        // This is designed to give reasonable stability for 1 or 2 orbits on the photon sphere of a Schwarzschild black hole of radius 1.
        dl = 1e-1 * (8./(deviation*deviation));
        if (dl > 5.)
        {
            // Too large; set to max parameter step.
            return 5.;
        }
        else
        {
            return dl;
        }
    }
}

// Advances a photon/pixel with an adaptive timestep using RK4.
__device__ void DeviceTraceTools::advance(float x[4], float v[4], float metric[4][4], float c_symbols[4][4][4], float metric_derivs[4][4][4])
{
    // Calculate parameter step.
    float dl { calculateParameterStep(metric) };
    // float dl { 0.1 };
    float mult_factor { dl/6.f };

    // Currently advances with RK4.
    // WARNING: Potential register spilling here; this requires a lot of memory.
    float x_step[4];
    float v_step[4];
    float x_temp[4];
    float v_temp[4];
    float k_n_minus_1_x[4];
    float k_n_x[4];
    float k_n_minus_1_v[4];
    float k_n_v[4];

    // Calculate k_1.
    getChristoffelSymbols(x, metric, c_symbols, metric_derivs);
    for (int i { 0 }; i < 4; i++)
    {
        k_n_x[i] = v[i];
        k_n_v[i] = 0;
        for (int j { 0 }; j < 4; j++)
        {
            #pragma unroll
            for (int k { 0 }; k < 4; k++)
            {
                k_n_v[i] -= c_symbols[i][j][k]*v[j]*v[k];
            }
        }
        x_step[i] = k_n_x[i];
        v_step[i] = k_n_v[i];
    }

    // Calculate k_2 and k_3.
    for (int u { 0 }; u < 2; u++)
    {
        #pragma unroll
        for (int i { 0 }; i < 4; i++)
        {
            k_n_minus_1_x[i] = k_n_x[i];
            k_n_minus_1_v[i] = k_n_v[i];
            x_temp[i] = x[i] + 0.5*dl*k_n_minus_1_x[i];
            v_temp[i] = v[i] + 0.5*dl*k_n_minus_1_v[i];
        }
        // Overwrite metric to avoid allocating another 16 floats.
        getMetricTensor(x_temp, metric);
        getChristoffelSymbols(x_temp, metric, c_symbols, metric_derivs);
        for (int i { 0 }; i < 4; i++)
        {
            k_n_x[i] = v[i] + 0.5*dl*k_n_minus_1_v[i];
            k_n_v[i] = 0;
            for (int j { 0 }; j < 4; j++)
            {
                #pragma unroll
                for (int k { 0 }; k < 4; k++)
                {
                    k_n_v[i] -= c_symbols[i][j][k]*v_temp[j]*v_temp[k];
                }
            }
            x_step[i] += 2.*k_n_x[i];
            v_step[i] += 2.*k_n_v[i];
        }
    }

    // Calculate k_4.
    #pragma unroll
    for (int i { 0 }; i < 4; i++)
    {
        k_n_minus_1_x[i] = k_n_x[i];
        k_n_minus_1_v[i] = k_n_v[i];
        x_temp[i] = x[i] + dl*k_n_minus_1_x[i];
        v_temp[i] = v[i] + dl*k_n_minus_1_v[i];
    }
    getMetricTensor(x_temp, metric);
    getChristoffelSymbols(x_temp, metric, c_symbols, metric_derivs);
    for (int i { 0 }; i < 4; i++)
    {
        k_n_x[i] = v[i] + dl*k_n_minus_1_v[i];
        k_n_v[i] = 0.;
        for (int j { 0 }; j < 4; j++)
        {
            #pragma unroll
            for (int k { 0 }; k < 4; k++)
            {
                k_n_v[i] -= c_symbols[i][j][k]*v_temp[j]*v_temp[k];
            }
        }
        x_step[i] += k_n_x[i];
        v_step[i] += k_n_v[i];

        // Advance x and v.
        x[i] += mult_factor*x_step[i];
        v[i] += mult_factor*v_step[i];
    }
    // Update metric to get it ready for the next step.
    getMetricTensor(x, metric);
}

// Gets a pointer to the RGB pixel from the sky map at pixel (x, y), where (0, 0) is the top-left pixel.
__device__ void DeviceTraceTools::readPixelFromSkyMap(unsigned char *pixel, unsigned char *device_sky_map, int &x, int &y, int &sky_pixels_w, int &byte_depth)
{
    pixel = &device_sky_map[(y*sky_pixels_w + x)*byte_depth];
}

// Run the actual raytracing loop. All the appropriate variables need to be assigned and defined before this can work (without undefined behaviour).
// TODO: Currently only defined for a camera outside the photon sphere of the Schwarzschild metric. Make this work for general metrics
// (i.e. some sort of event horizon-detector to terminate a photon?).
__global__ void traceImage(unsigned char *device_sky_map, unsigned char *device_camera_pixel_array)
{
    // This is currently intended for 8x4 thread blocks.
    // TODO: For now, the Christoffel symbols are in shared memory and will use 8 KiB. Test later
    // whether they can be moved to registers without spilling for a significant speed boost.
    // Same thing for the metric derivative components.
    __shared__ float c_symbols[8][4][4][4][4];
    __shared__ float metric_derivs[8][4][4][4][4];

    int pixel_x = blockIdx.x*blockDim.x + threadIdx.x;
    int pixel_y = blockIdx.y*blockDim.y + threadIdx.y;
    int pixel_index = 3*(pixel_y*DeviceTraceTools::device_pixels_w + pixel_x);
    bool consumed = false;

    if (pixel_x < DeviceTraceTools::device_pixels_w && pixel_y < DeviceTraceTools::device_pixels_h)
    {
        float x[4];
        float v[4];
        float metric[4][4];
        #pragma unroll
        for (int i { 0 }; i < 4; i++)
        {
            x[i] = DeviceTraceTools::device_camera_coords[i];
        }

        DeviceTraceTools::getMetricTensor(x, metric);
        DeviceTraceTools::calculateStartVelocity(pixel_x, pixel_y, v, metric);
        DeviceTraceTools::normaliseV(v);

        // Set to true if the photon enters the photon sphere.
        float sky_dist_squared = DeviceTraceTools::device_sky_map_distance_squared;
        float dist_squared = x[1]*x[1] + x[2]*x[2] + x[3]*x[3];
        while (dist_squared < sky_dist_squared)
        {
            if (dist_squared < 2.25)
            {
                // Entered the photon radius if true; terminate the photon.
                consumed = true;
                break;
            }
            // Otherwise, advance the simulation with RK4.
            DeviceTraceTools::advance(x, v, metric, &c_symbols[threadIdx.x][threadIdx.y][0], &metric_derivs[threadIdx.x][threadIdx.y][0]);
            dist_squared = x[1]*x[1] + x[2]*x[2] + x[3]*x[3];
        }
    }

    if (consumed == true)
    {
        // Entered the photon sphere; set to black.
        device_camera_pixel_array[pixel_index] = 0;
        device_camera_pixel_array[pixel_index+1] = 0;
        device_camera_pixel_array[pixel_index+2] = 0;
    }
    else
    {
        device_camera_pixel_array[pixel_index] = 150;
        device_camera_pixel_array[pixel_index+1] = 150;
        device_camera_pixel_array[pixel_index+2] = 150;
    }
}

// Calculates the Hamilton (quaternionic) product of u with v.
__device__ void hamiltonProduct(float u[4], float v[4], float result[4])
{
    result[0] = u[0]*v[0] - (u[1]*v[1] + u[2]*v[2] + u[3]*v[3]);
    // Cross product of the vector components of u and v is needed.
    float cross[3];
    cross[0] = u[2]*v[3] - u[3]*v[2];
    cross[1] = u[3]*v[1] - u[1]*v[3];
    cross[2] = u[1]*v[2] - u[2]*v[1];
    for (int i { 1 }; i < 4; i++)
    {
        result[i] = u[0]*v[i] + v[0]*u[i] + cross[i-1];
    }
}

// Rotates a 3D cartesian vector, vec (given as a quaternion with no real part), by the given quaternion, rotation_quat.
// result will be the rotated vector represented as a quaternion with no real part.
__device__ void rotateVecByQuat(float vec[4], float rotation_quat[4], float result[4])
{
    // Assume that rotation_quat is normalised.
    float rotation_quat_inverse[4];
    rotation_quat_inverse[0] = rotation_quat[0];
    rotation_quat_inverse[1] = -rotation_quat[1];
    rotation_quat_inverse[2] = -rotation_quat[2];
    rotation_quat_inverse[3] = -rotation_quat[3];
    float intermediate_result[4];
    hamiltonProduct(vec, rotation_quat_inverse, intermediate_result);
    hamiltonProduct(rotation_quat, intermediate_result, result);
}
